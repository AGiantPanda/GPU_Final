#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <stdexcept>
#include <vector>

// #include <opencv2/opencv.hpp>
#include "opencv2/core.hpp"
#include "opencv2/core/utility.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudafeatures2d.hpp"
#include "opencv2/opencv_modules.hpp"

#include "opencv2/stitching/detail/blenders.hpp"

//#include "opencv2/gpu.hpp"
#include "opencv2/xfeatures2d/cuda.hpp"

#include "warp.h"
#include "../utils/SyncedMemory.h"
#include "../utils/Timer.h"

using namespace std;
using namespace cv;
using namespace cv::detail;
using namespace cv::cuda;

typedef Vec<float, 9> Vec9f;
typedef Vec<double, 9> Vec9d;
typedef vector<vector<Mat> > BundleHomo;
#define index(i, j) (j)*width+(i)
#define cellind(i, j) (j)*cellwidth+(i)

warp::warp(asapWarp asap)
{
	this->asap = asap;
	this->height = asap.height;
	this->width = asap.width;
	this->cellheight = this->height-1;
	this->cellwidth = this->width-1;

	vector<Point2f> cellPtsT(width*height);
	vector<Point2f> cellPts0(width*height);
	
	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			cellPtsT[index(i, j)] = asap.compute_pos(i, j);
			cellPts0[index(i, j)] = asap.compute_pos(i, j);
		}

	this->cellPtsT = cellPtsT;
	this->cellPts0 = cellPts0;
}

warp::~warp(){}

__host__ __device__ int findCell(const float* point, const float x, const float y, const int pointNum, 
						const int cellwidth, const int cellheight)
{
	float minx, maxx, miny, maxy;
	minx = maxx = point[0];
	miny = maxy = point[1];
	int i, j;
	for (i = 1; i < pointNum; i++)
	{
		minx = point[i*2+0] < minx ? point[i*2+0] : minx;
		maxx = point[i*2+0] > maxx ? point[i*2+0] : maxx;
		miny = point[i*2+1] < miny ? point[i*2+1] : miny;
		maxy = point[i*2+1] > maxy ? point[i*2+1] : maxy;
	}

	if (x > maxx || x < minx || y > maxy || y < miny)
		return -1;
	
	float V00x, V00y, V10x, V10y, V01x, V01y, V11x, V11y;
	float ax, ay, bx, by;
	for (i = 0; i < cellwidth; i++)
		for (j = 0; j < cellheight; j++)
		{

			V00x = point[2*(j*(cellwidth+1)+i)+0];
			V00y = point[2*(j*(cellwidth+1)+i)+1];
			V10x = point[2*(j*(cellwidth+1)+i+1)+0];
			V10y = point[2*(j*(cellwidth+1)+i+1)+1];
			V01x = point[2*((j+1)*(cellwidth+1)+i)+0];
			V01y = point[2*((j+1)*(cellwidth+1)+i)+1];
			V11x = point[2*((j+1)*(cellwidth+1)+i+1)+0];
			V11y = point[2*((j+1)*(cellwidth+1)+i+1)+1];

			minx = V00x < V01x ? V00x : V01x;
			maxx = V10x > V11x ? V10x : V11x;
			miny = V00y < V10y ? V00y : V10y;
			maxy = V01y > V11y ? V01y : V11y;

			if (x > maxx || x < minx || y > maxy || y < miny)
				continue;
		
			ax = V10x-V00x;
			ay = V10y-V00y;
			bx = x-V00x;
			by = y-V00y;
			if (ax*by-ay*bx < 0)
				continue;

			ax = V11x-V10x;
			ay = V11y-V10y;
			bx = x-V10x;
			by = y-V10y;
			if (ax*by-ay*bx < 0)
				continue;

			ax = V01x-V11x;
			ay = V01y-V11y;
			bx = x-V11x;
			by = y-V11y;
			if (ax*by-ay*bx < 0)
				continue;

			ax = V00x-V01x;
			ay = V00y-V01y;
			bx = x-V01x;
			by = y-V01y;
			if (ax*by-ay*bx < 0)
				continue;
			
			return j*cellwidth+i;
		}

	return -1;
}

__global__ void warpImgByVertexGPU(PtrStepSz<uchar3> const img, PtrStepSz<uchar3> warpimg, 
						const float* ptT, const float* warppt0, const float* Phinv, const float* CH, 
						const int cellwidth, const int cellheight, const int pointNum)
{
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float shared_ptT[192];
	__shared__ float shared_warppt0[192];
	int idx;
	for (int i = 0; i < pointNum / (blockDim.x * blockDim.y) + 1; i++)
	{
		idx = 2*((blockDim.x * blockDim.y)*i + threadIdx.y * blockDim.x + threadIdx.x);
		if (idx < pointNum*2)
		{
			shared_ptT[idx] = ptT[idx];
			shared_ptT[idx+1] = ptT[idx+1];
			shared_warppt0[idx] = warppt0[idx];
			shared_warppt0[idx+1] = warppt0[idx+1];
		}	
	}

	__shared__ float shared_Phinv[576];
	__shared__ float shared_CH[576];
	idx = threadIdx.y * blockDim.x + threadIdx.x;
	if (idx < 576)
	{
		shared_Phinv[idx] = Phinv[idx];
		shared_CH[idx] = CH[idx];
	}

	__syncthreads();

	if (x < img.cols && y <= img.rows)
	{
		float ptx = float(x);
		float pty = float(y);
		int cellindex = findCell(shared_ptT, ptx, pty, pointNum, cellwidth, cellheight);

		if (cellindex < 0)
			return;

		float warpTx = shared_Phinv[9*cellindex+0]*ptx + shared_Phinv[9*cellindex+1]*pty + shared_Phinv[9*cellindex+2]*1.f;
		float warpTy = shared_Phinv[9*cellindex+3]*ptx + shared_Phinv[9*cellindex+4]*pty + shared_Phinv[9*cellindex+5]*1.f;
		float warpTz = shared_Phinv[9*cellindex+6]*ptx + shared_Phinv[9*cellindex+7]*pty + shared_Phinv[9*cellindex+8]*1.f;
		warpTx = warpTx / warpTz;
		warpTy = warpTy / warpTz;

		cellindex = findCell(shared_warppt0, warpTx, warpTy, pointNum, cellwidth, cellheight);

		if (cellindex < 0)
			return;
		float warpTx2 = shared_CH[9*cellindex+0]*warpTx + shared_CH[9*cellindex+1]*warpTy + shared_CH[9*cellindex+2]*1.f;
		float warpTy2 = shared_CH[9*cellindex+3]*warpTx + shared_CH[9*cellindex+4]*warpTy + shared_CH[9*cellindex+5]*1.f;
		float warpTz2 = shared_CH[9*cellindex+6]*warpTx + shared_CH[9*cellindex+7]*warpTy + shared_CH[9*cellindex+8]*1.f;
		warpTx2 = warpTx2 / warpTz2;
		warpTy2 = warpTy2 / warpTz2;

		int floorx = int(warpTx2);
		int floory = int(warpTy2);
		float deltax = warpTx2-float(floorx);
		float deltay = warpTy2-float(floory);

		if (floorx > img.cols-1 || floorx+1 < 0 || floory > img.rows-1 || floory+1 < 0)
			return;
		
		uchar3 color00 = img(floory, floorx);
		uchar3 color10 = img(floory+1, floorx);
		uchar3 color01 = img(floory, floorx+1);
		uchar3 color11 = img(floory+1, floorx+1);

		float b = (1-deltax)*(1-deltay)*float(color00.x) 
				+ (1-deltax)*(deltay)  *float(color10.x)
				+ (deltax)*(deltay)    *float(color11.x)
				+ (deltax)*(1-deltay)  *float(color01.x);
		float g = (1-deltax)*(1-deltay)*float(color00.y) 
				+ (1-deltax)*(deltay)  *float(color10.y)
				+ (deltax)*(deltay)    *float(color11.y)
				+ (deltax)*(1-deltay)  *float(color01.y);
		float r = (1-deltax)*(1-deltay)*float(color00.z) 
				+ (1-deltax)*(deltay)  *float(color10.z)
				+ (deltax)*(deltay)    *float(color11.z)
				+ (deltax)*(1-deltay)  *float(color01.z);
		b = b > 255 ? 255 : b;
		b = b < 0 ? 0 : b;
		g = g > 255 ? 255 : g;
		g = g < 0 ? 0 : g;
		r = r > 255 ? 255 : r;
		r = r < 0 ? 0 : r;
		
		warpimg(y, x).x = uchar(b);
		warpimg(y, x).y = uchar(g);
		warpimg(y, x).z = uchar(r);
		
	}
}

vector<Point2i> warp::compute_corner(vector<Point2f> warpPts0, float *Pinv)
{
	vector<Point2i> corner;
	Point2f V00 = warpPts0[index(0, 0)];
	Point2f V01 = warpPts0[index(0, height-1)];
	Point2f V10 = warpPts0[index(width-1, 0)];
	Point2f V11 = warpPts0[index(width-1, height-1)];

	Mat Pinv00(3, 3, CV_32F);
	Mat Pinv01(3, 3, CV_32F);
	Mat Pinv10(3, 3, CV_32F);
	Mat Pinv11(3, 3, CV_32F);	

	int ind = cellind(0, 0);
	Pinv00.at<float>(0, 0) = Pinv[9*ind+0];
	Pinv00.at<float>(0, 1) = Pinv[9*ind+1];
	Pinv00.at<float>(0, 2) = Pinv[9*ind+2];
	Pinv00.at<float>(1, 0) = Pinv[9*ind+3];
	Pinv00.at<float>(1, 1) = Pinv[9*ind+4];
	Pinv00.at<float>(1, 2) = Pinv[9*ind+5];
	Pinv00.at<float>(2, 0) = Pinv[9*ind+6];
	Pinv00.at<float>(2, 1) = Pinv[9*ind+7];
	Pinv00.at<float>(2, 2) = Pinv[9*ind+8];

	ind = cellind(cellwidth-1, 0);
	Pinv10.at<float>(0, 0) = Pinv[9*ind+0];
	Pinv10.at<float>(0, 1) = Pinv[9*ind+1];
	Pinv10.at<float>(0, 2) = Pinv[9*ind+2];
	Pinv10.at<float>(1, 0) = Pinv[9*ind+3];
	Pinv10.at<float>(1, 1) = Pinv[9*ind+4];
	Pinv10.at<float>(1, 2) = Pinv[9*ind+5];
	Pinv10.at<float>(2, 0) = Pinv[9*ind+6];
	Pinv10.at<float>(2, 1) = Pinv[9*ind+7];
	Pinv10.at<float>(2, 2) = Pinv[9*ind+8];

	ind = cellind(0, cellheight-1);
	Pinv01.at<float>(0, 0) = Pinv[9*ind+0];
	Pinv01.at<float>(0, 1) = Pinv[9*ind+1];
	Pinv01.at<float>(0, 2) = Pinv[9*ind+2];
	Pinv01.at<float>(1, 0) = Pinv[9*ind+3];
	Pinv01.at<float>(1, 1) = Pinv[9*ind+4];
	Pinv01.at<float>(1, 2) = Pinv[9*ind+5];
	Pinv01.at<float>(2, 0) = Pinv[9*ind+6];
	Pinv01.at<float>(2, 1) = Pinv[9*ind+7];
	Pinv01.at<float>(2, 2) = Pinv[9*ind+8];

	ind = cellind(cellwidth-1, cellheight-1);
	Pinv11.at<float>(0, 0) = Pinv[9*ind+0];
	Pinv11.at<float>(0, 1) = Pinv[9*ind+1];
	Pinv11.at<float>(0, 2) = Pinv[9*ind+2];
	Pinv11.at<float>(1, 0) = Pinv[9*ind+3];
	Pinv11.at<float>(1, 1) = Pinv[9*ind+4];
	Pinv11.at<float>(1, 2) = Pinv[9*ind+5];
	Pinv11.at<float>(2, 0) = Pinv[9*ind+6];
	Pinv11.at<float>(2, 1) = Pinv[9*ind+7];
	Pinv11.at<float>(2, 2) = Pinv[9*ind+8];

	vector<Point2f> pt(1);
	vector<Point2f> warppt(1);
	pt[0] = V00;
	perspectiveTransform(pt, warppt, Pinv00.inv());
	corner.push_back(Point2i(warppt[0]));

	pt[0] = V01;
	perspectiveTransform(pt, warppt, Pinv01.inv());
	corner.push_back(Point2i(warppt[0]));

	pt[0] = V10;
	perspectiveTransform(pt, warppt, Pinv10.inv());
	corner.push_back(Point2i(warppt[0]));

	pt[0] = V11;
	perspectiveTransform(pt, warppt, Pinv11.inv());
	corner.push_back(Point2i(warppt[0]));

	return corner;
}

void warp::findCut(Mat img, int* cutxy, vector<Point2i> corner)
{
	int minx = min(max(max(corner[0].x, corner[1].x), 0), img.cols/3);
	int miny = min(max(max(corner[0].y, corner[2].y), 0), img.rows/3);
	int maxx = max(min(min(corner[2].x, corner[3].x), img.cols-1), 2*img.cols/3);
	int maxy = max(min(min(corner[1].y, corner[3].y), img.rows-1), 2*img.rows/3);
	uchar zero = uchar(0);

	for (int x = minx; x <= maxx; x++)
	{
		for (int y = miny; y < miny+10; y++)
		{
			//printf("img.at<uchar3>(%d, %d) = (%u, %u, %u)\n", y, x, img.at<uchar3>(y, x).x, img.at<uchar3>(y, x).y, img.at<uchar3>(y, x).z);
			if (img.at<uchar3>(y, x).x != zero || img.at<uchar3>(y, x).y != zero || img.at<uchar3>(y, x).z != zero)
			{
				if (miny < y)
					miny = y;
				break;
			}
		}
	}
	for (int x = minx; x <= maxx; x++)
	{
		for (int y = maxy; y > maxy-10; y--)
		{
			//printf("img.at<uchar3>(%d, %d) = (%u, %u, %u)\n", y, x, img.at<uchar3>(y, x).x, img.at<uchar3>(y, x).y, img.at<uchar3>(y, x).z);
			if (img.at<uchar3>(y, x).x != zero || img.at<uchar3>(y, x).y != zero || img.at<uchar3>(y, x).z != zero)
			{
				
				if (maxy > y)
					maxy = y;
				break;
			}
		}
	}

	for (int y = miny; y <= maxy; y++)
	{
		for (int x = minx; x < minx+10; x++)
		{
			if (img.at<uchar3>(y, x).x != zero || img.at<uchar3>(y, x).y != zero || img.at<uchar3>(y, x).z != zero)
			{
				if (minx < x)
					minx = x;
				break;
			}
		}
	}
	for (int y = miny; y <= maxy; y++)
	{
		for (int x = maxx; x > maxx-10; x--)
		{
			if (img.at<uchar3>(y, x).x != zero || img.at<uchar3>(y, x).y != zero || img.at<uchar3>(y, x).z != zero)
			{
				if (maxx > x)
					maxx = x;
				break;
			}
		}
	}

	cutxy[0] = minx;
	cutxy[1] = maxx;
	cutxy[2] = miny;
	cutxy[3] = maxy;

	/*
	cout << "minx = " << minx << endl;
	cout << "maxx = " << maxx << endl;
	cout << "miny = " << miny << endl;
	cout << "maxy = " << maxy << endl;
	//*/

	// resize(img(Rect(minx, miny, sizex, sizey)), cutimg, img.size());
	return;
}

void warp::compute_homo(float *C, const vector<Point2f> &pts, const vector<Point2f> &warpPts)
{
	for (int i = 0; i < width-1; i++)
		for (int j = 0; j < height-1; j++)
		{
			int ind = 9*(j*(width-1)+i);
			vector<Point2f> P, WP;
			for (int I = i; I < i+2; I++)
				for (int J = j; J < j+2; J++)
				{
					P.push_back(pts[index(I, J)]);
					WP.push_back(warpPts[index(I, J)]);
				}

			Mat h = findHomography(P, WP);
			C[ind+0] = float(h.at<double>(0, 0));
			C[ind+1] = float(h.at<double>(0, 1));
			C[ind+2] = float(h.at<double>(0, 2));
			C[ind+3] = float(h.at<double>(1, 0));
			C[ind+4] = float(h.at<double>(1, 1));
			C[ind+5] = float(h.at<double>(1, 2));
			C[ind+6] = float(h.at<double>(2, 0));
			C[ind+7] = float(h.at<double>(2, 1));
			C[ind+8] = float(h.at<double>(2, 2));
		}
}

void warp::warpImageMeshbyVertexGPU(Mat img, Mat & warpimg, vector<Point2f> warpPts0, vector<Point2f> warpPtsT, int * cutxy)
{
	float *Pinv = new float[(width-1)*(height-1)*9];
 	compute_homo(Pinv, cellPtsT, warpPtsT);
	float *C = new float[(width-1)*(height-1)*9];
	compute_homo(C, warpPts0, cellPts0);

	int N = cellPtsT.size();
	float *ptrT  = new float[N*2];
	float *warp0 = new float[N*2];
	for (int i = 0; i < N; i++)
	{
		ptrT[i*2+0] = cellPtsT[i].x;
		ptrT[i*2+1] = cellPtsT[i].y;
		warp0[i*2+0] = warpPts0[i].x;
		warp0[i*2+1] = warpPts0[i].y;
	}
	
	float *ptrT_mat, *warp0_mat;
	hipMalloc((void **) &ptrT_mat,   N*2*sizeof(float));
	hipMalloc((void **) &warp0_mat, N*2*sizeof(float));
	hipMemcpy(ptrT_mat,   ptrT,  N*2*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(warp0_mat, warp0, N*2*sizeof(float), hipMemcpyHostToDevice);

	int M = (width-1)*(height-1);
	float *Pinv_device, *C_device;
	hipMalloc((void **) &Pinv_device, M*9*sizeof(float));
	hipMalloc((void **) &C_device,    M*9*sizeof(float));
	hipMemcpy(Pinv_device, Pinv,  M*9*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C_device,    C,     M*9*sizeof(float), hipMemcpyHostToDevice);

	const int BLOCK_WIDTH = 32;
	const int BLOCK_HEIGHT = 32;
	const int X_BLOCK_NUM = (img.cols-1)/BLOCK_WIDTH + 1;
	const int Y_BLOCK_NUM = (img.rows-1)/BLOCK_HEIGHT + 1;
	dim3 block = dim3(X_BLOCK_NUM, Y_BLOCK_NUM);
	dim3 thread = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);

	GpuMat img_device; 
	img_device.upload(img);
	GpuMat warpimg_device(img.size(), CV_8UC3);
	warpimg_device.setTo(0);


	//warpImgByVertexGPU(PtrStepSz<uchar3> const img, PtrStepSz<uchar3> warpimg, 
	//					const float* pt, const float* warppt0, const float* PHinv, const float* C,
	//					const int cellwidth, const int cellheight, const int pointNum)
	if (N != 81)
	{
		cerr << "pointNum must equal to the given number 81 in the cuda code, need to modify the cuda code" << endl;
		return;
	}

	warpImgByVertexGPU<<< block, thread >>>(img_device, warpimg_device, ptrT_mat, warp0_mat, 
											Pinv_device, C_device, width-1, height-1, N);
	warpimg_device.download(warpimg);

	// minx, maxx, miny, maxy

	// void findCut(float *cutxy, float* warpT, float *warp0, float *Pinv, float *C, const int cellwidth, const int cellheight, const int pointNum)
	vector<Point2i> corner = compute_corner(warpPts0, Pinv);
	findCut(warpimg, cutxy, corner);
	for (int i = 0; i < 4; i++)
		cout << "corner = " << corner[i] << endl;

	cout << "cutx = " << cutxy[0] << ", " << cutxy[1] << endl;
	cout << "cuty = " << cutxy[2] << ", " << cutxy[3] << endl;

	/* imshow
	namedWindow("warpimg", WINDOW_AUTOSIZE);
	imshow("warpimg", warpimg);
	waitKey(0);
	//*/

	return;
}

void warp::warpImageMeshGPU(Mat img, Mat & warpimg, BundleHomo Cpath, BundleHomo P)
{
	vector<Point2f> cellPtsT(width*height);
	vector<Point2f> cellPts0(width*height);
	vector<Point2f> warpPtsT(width*height);
	vector<Point2f> warpPts0(width*height);
	
	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			cellPtsT[index(i, j)] = asap.compute_pos(i, j);
			cellPts0[index(i, j)] = asap.compute_pos(i, j);
		}

	vector<Point2f> pt(1);
	vector<Point2f> warpPt(1);

	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			int N = 0;
			Point2f sumpt(0.f, 0.f);
			pt[0] = cellPts0[index(i, j)];
			for (int I = max(0, i-1); I < min(width-1, i+1); I++)
				for (int J = max(0, j-1); J < min(height-1, j+1); J++)
				{
					N++;
					perspectiveTransform(pt, warpPt, Cpath[I][J].inv());
					sumpt += warpPt[0];
				}

			warpPts0[index(i, j)] = sumpt / N;
 		}

 	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			int N = 0;
			Point2f sumpt(0.f, 0.f);
			pt[0] = cellPtsT[index(i, j)];
			for (int I = max(0, i-1); I < min(width-1, i+1); I++)
				for (int J = max(0, j-1); J < min(height-1, j+1); J++)
				{
					N++;
					perspectiveTransform(pt, warpPt, P[I][J].inv());
					sumpt += warpPt[0];
				}

			warpPtsT[index(i, j)] = sumpt / N;		
		}

	float *Pinv = new float[(width-1)*(height-1)*9];
 	compute_homo(Pinv, cellPtsT, warpPtsT);
	float *C = new float[(width-1)*(height-1)*9];
	compute_homo(C, warpPts0, cellPts0);

	int N = cellPtsT.size();
	float *ptrT  = new float[N*2];
	float *warp0 = new float[N*2];
	for (int i = 0; i < N; i++)
	{
		ptrT[i*2+0] = cellPtsT[i].x;
		ptrT[i*2+1] = cellPtsT[i].y;
		warp0[i*2+0] = warpPts0[i].x;
		warp0[i*2+1] = warpPts0[i].y;
	}
	
	float *ptrT_mat, *warp0_mat;
	hipMalloc((void **) &ptrT_mat,   N*2*sizeof(float));
	hipMalloc((void **) &warp0_mat, N*2*sizeof(float));
	hipMemcpy(ptrT_mat,   ptrT,  N*2*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(warp0_mat, warp0, N*2*sizeof(float), hipMemcpyHostToDevice);

	int M = (width-1)*(height-1);
	float *Pinv_device, *C_device;
	hipMalloc((void **) &Pinv_device, M*9*sizeof(float));
	hipMalloc((void **) &C_device,    M*9*sizeof(float));
	hipMemcpy(Pinv_device, Pinv,  M*9*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C_device,    C,     M*9*sizeof(float), hipMemcpyHostToDevice);


	const int BLOCK_WIDTH = 16;
	const int BLOCK_HEIGHT = 12;
	const int X_BLOCK_NUM = (img.cols-1)/BLOCK_WIDTH + 1;
	const int Y_BLOCK_NUM = (img.rows-1)/BLOCK_HEIGHT + 1;
	dim3 block = dim3(X_BLOCK_NUM, Y_BLOCK_NUM);
	dim3 thread = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);

	GpuMat img_device; 
	img_device.upload(img);
	GpuMat warpimg_device(img.size(), CV_8UC3);
	warpimg_device.setTo(0);


	//warpImgByVertexGPU(PtrStepSz<uchar3> const img, PtrStepSz<uchar3> warpimg, 
	//					const float* pt, const float* warppt0, const float* PHinv, const float* C,
	//					const int cellwidth, const int cellheight, const int pointNum)
	warpImgByVertexGPU<<< block, thread >>>(img_device, warpimg_device, ptrT_mat, warp0_mat, 
											Pinv_device, C_device, width-1, height-1, N);
	warpimg_device.download(warpimg);

	/* imshow
	namedWindow("warpimg", WINDOW_AUTOSIZE);
	imshow("warpimg", warpimg);
	waitKey(0);
	//*/

	return;
}

void warp::warpImageMesh(Mat img, Mat & warpimg, BundleHomo C, BundleHomo P)
{
	vector<Point2f> cellPtsT(width*height);
	vector<Point2f> cellPts0(width*height);
	
	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			cellPtsT[index(i, j)] = asap.compute_pos(i, j);
		}

	vector<Point2f> pt(1);
	vector<Point2f> warpPt(1);

	float minx, miny, maxx, maxy;
	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			int N = 0;
			Point2f sumpt(0.f, 0.f);
			pt[0] = cellPtsT[index(i, j)];
//cerr << "(" << i << ", " << j << ")" << endl;
			for (int I = max(0, i-1); I < min(width-1, i+1); I++)
				for (int J = max(0, j-1); J < min(height-1, j+1); J++)
				{
					N++;
					perspectiveTransform(pt, warpPt, C[I][J].inv());
//cerr << "\t(" << I << ", " << J << ") " << endl;
//cerr << "C[I][J].inv() = " << C[I][J].inv() << endl;
//cerr << "warpPt = " << warpPt[0] << endl;
					sumpt += warpPt[0];
				}

			cellPts0[index(i, j)] = sumpt / N;
//cerr << "(" << i << ", " << j << ") = " << cellPts0[index(i, j)] << endl;
			if (i == 0 && j == 0)
			{
				minx = cellPts0[index(i, j)].x;
				maxx = cellPts0[index(i, j)].x;
				miny = cellPts0[index(i, j)].y;
				maxy = cellPts0[index(i, j)].y;
			}
			else
			{
				minx = min(cellPts0[index(i, j)].x, minx);
				maxx = max(cellPts0[index(i, j)].x, maxx);
				miny = min(cellPts0[index(i, j)].y, miny);
				maxy = max(cellPts0[index(i, j)].y, maxy);
			}

 		}

	float dist = max(maxx-minx, maxy-miny); 
	float rate = 1.f;
cerr << "dist = " << dist << endl;
	if (dist > 4000.f)
	{
		rate = 4000.f/dist;
		for (int i = 0; i < cellPts0.size(); i++)
			cellPts0[i] = cellPts0[i] * rate;
	}

	Mat originImg;
	Point2f offset = warpImgByVertex(img, originImg, cellPtsT, cellPts0, true);
	/* imshow
	namedWindow("originImg", WINDOW_AUTOSIZE);
	imshow("originImg", originImg);
	waitKey(0);
	//*/

	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
		{
			int N = 0;
			Point2f sumpt(0.f, 0.f);
			pt[0] = cellPtsT[index(i, j)];
			for (int I = max(0, i-1); I < min(width-1, i+1); I++)
				for (int J = max(0, j-1); J < min(height-1, j+1); J++)
				{
					N++;
					perspectiveTransform(pt, warpPt, P[I][J].inv());
					sumpt += warpPt[0];
				}

			cellPts0[index(i, j)] = sumpt / N;
////cerr << "(" << i << ", " << j << ") = " << cellPts0[index(i, j)] << endl;			
		}

	if (dist > 4000.f)
	{
		rate = 4000.f/dist;
		for (int i = 0; i < cellPts0.size(); i++)
			cellPts0[i] = cellPts0[i] * rate;
	}

	warpImgByVertex(originImg, warpimg, cellPts0, cellPtsT, false, offset, img.size());
	originImg.release();

	/* imshow
	namedWindow("warpimg", WINDOW_AUTOSIZE);
	imshow("warpimg", warpimg);
	waitKey(0);
	//*/

}


Point warp::warpImgByVertex(Mat img, Mat & warpimg, vector<Point2f> pt, vector<Point2f> warppt, 
						bool all, Point offset, Size s)
{
	// offset means that offset -> (0, 0) 

	int minx = img.size().width;
	int maxx = 0;
	int miny = img.size().height;
	int maxy = 0;
	for (int i = 0; i < pt.size(); i++)
	{
		pt[i].x += offset.x;
		pt[i].y += offset.y;
		warppt[i].x += offset.x;
		warppt[i].y += offset.y;

		minx = min(int(floor(warppt[i].x)), minx);
		miny = min(int(floor(warppt[i].y)), miny);
		maxx = max(int(ceil(warppt[i].x)), maxx);
		maxy = max(int(ceil(warppt[i].y)), maxy);
	}
	offset.x -= minx;
	offset.y -= miny;

	for (int i = 0; i < pt.size(); i++)
	{
		warppt[i].x -= minx;
		warppt[i].y -= miny;
	}
	maxx -= minx;
	minx -= minx;
	maxy -= miny;
	miny -= miny;
	
	int sizex = maxx - minx + 1;
	int sizey = maxy - miny + 1;

	FeatherBlender blender(0.5f);  //sharpness
	blender.prepare(Rect(0, 0, sizex, sizey));

	Mat frame;
	img.convertTo(frame, CV_16SC3);

	for (int i = 0; i < width-1; i++)
		for (int j = 0; j < height-1; j++)
		{
			Mat warp_frame, mask, h;
			mask = Mat::zeros(Size(sizex, sizey), CV_8U);

			Point countour[1][4];
			countour[0][0] = Point(warppt[index(i, j)]);
			countour[0][1] = Point(warppt[index(i, j+1)]);
			countour[0][2] = Point(warppt[index(i+1, j+1)]);
			countour[0][3] = Point(warppt[index(i+1, j)]);			

			const Point* ppt[1] = { countour[0] }; 
			int npt[] = {4};  
			fillPoly(mask, ppt, npt, 1, Scalar::all(255)); 
			polylines(mask, ppt, npt, 1, 1, Scalar::all(255), 10);  

			vector<Point2f> P, WP;
			for (int I = i; I < i+2; I++)
				for (int J = j; J < j+2; J++)
				{
					P.push_back(pt[index(I, J)]);
					WP.push_back(warppt[index(I, J)]);
				}

			h = findHomography(P, WP);
////cerr << "(" << i << ", " << j << ") = " << h << endl;	
			warpPerspective(frame, warp_frame, h, mask.size());
			// fb->feed(warp_frame, mask, Point(0, 0));
			blender.feed(warp_frame, mask, Point(0, 0));
		}

	Mat mask = Mat::zeros(Size(sizex, sizey), CV_8U);
	if (!all)
	{
		minx = offset.x;
		maxx = min(offset.x + s.width - 1, s.width- 1);
		miny = offset.y;
		maxy = min(offset.y + s.height - 1, s.height - 1);
	}
	else
	{
		s.width = sizex;
		s.height = sizey;
	}
	sizex = maxx - minx + 1;
	sizey = maxy - miny + 1;

	Mat warp_frame;

	mask(Rect(minx, miny, sizex, sizey)).setTo(Scalar::all(255));
	// fb->blend(warp_frame, mask);
	blender.blend(warp_frame, mask);
	//free(fb);
	warp_frame.convertTo(warp_frame, CV_8UC3);

	///* Draw Points on warpframe
	Mat warp_frame_points;
	DrawPoints(warp_frame, warp_frame_points, warppt, Point(0, 0));
	namedWindow("warp_frame_points", WINDOW_NORMAL);
	imshow("warp_frame_points", warp_frame_points);
	waitKey(0);
	//*/

	warpimg = Mat::zeros(s, warp_frame.type());
	warp_frame(Rect(minx, miny, sizex, sizey)).copyTo(warpimg(Rect(0, 0, sizex, sizey)));

	frame.release();
	warp_frame.release();
	mask.release();

	return offset;
}

void warp::DrawPoints(Mat img, Mat & pointImg, vector<Point2f> pts, Point offset)
{
	pointImg = img.clone();
	for (int i = 0; i < pts.size(); i++)
	{
		circle(pointImg, Point(pts[i])+offset, 3, Scalar(0, 0, 255), -1);
	}
}

#undef index
#undef cellind