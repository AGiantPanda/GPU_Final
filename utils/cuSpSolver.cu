#include "./cuSpSolver.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipsolver.h"

using namespace std;

bool cuSpSolver(int* ARow, int* ACol, float* AVal, float* B, float* X, int m, int nnz)
{
	hipsolverStatus_t solver_status;
	hipsparseStatus_t sparse_status;
	hipsolverSpHandle_t solver_handle = 0;
	hipsparseHandle_t sparse_handle = 0;
	hipsparseMatDescr_t descr = 0;

	// initialize
	sparse_status = hipsparseCreate(&sparse_handle);
	if(sparse_status != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreate failed" << endl;
		return 2;
	}
	sparse_status = hipsparseCreateMatDescr(&descr);
	if(sparse_status != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreateMatDescr failed" << endl;
		return 2;
	}
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	solver_status = hipsolverSpCreate(&solver_handle);
	if(solver_status != HIPSOLVER_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreate failed" << endl;
		return 2;
	}

	int* csrRow = 0; hipMalloc((void**)&csrRow, (m+1)*sizeof(csrRow[0]));
	int* cooRow = 0; hipMalloc((void**)&cooRow, nnz*sizeof(cooRow[0]));
	int* csrCol = 0; hipMalloc((void**)&csrCol, nnz*sizeof(csrCol[0]));
	float* csrVal = 0; hipMalloc((void**)&csrVal, nnz*sizeof(csrVal[0]));
	float* b = 0; hipMalloc((void**)&b, m*sizeof(b[0]));
	float* x = 0; hipMalloc((void**)&x, m*sizeof(x[0]));

	hipMemcpy(cooRow, ARow, (size_t)(nnz*sizeof(cooRow[0])), hipMemcpyHostToDevice);
	hipMemcpy(csrCol, ACol, (size_t)(nnz*sizeof(csrCol[0])), hipMemcpyHostToDevice);
	hipMemcpy(csrVal, AVal, (size_t)(nnz*sizeof(csrVal[0])), hipMemcpyHostToDevice);
	hipMemcpy(b, B, (size_t)(m*sizeof(b[0])), hipMemcpyHostToDevice);
	hipsparseXcoo2csr(sparse_handle, cooRow, nnz, m, csrRow, HIPSPARSE_INDEX_BASE_ZERO);

	int singular;
	solver_status = hipsolverSpScsrlsvqr(
		solver_handle,
		m,
		nnz,
		descr,
		csrVal,
		csrRow,
		csrCol,
		b,
		0.0,
		0,
		x,
		&singular);
	if(solver_status != HIPSOLVER_STATUS_SUCCESS)
	{
		cout<<"solve failed"<<endl;
		return 1;
	}
	else
	{
		hipMemcpy(X, x, (size_t)(m*sizeof(X[0])), hipMemcpyDeviceToHost);
	}

	hipFree(csrRow);
	hipFree(cooRow);
	hipFree(csrVal);
	hipFree(csrCol);
	hipFree(x);
	hipFree(b);
	hipsolverSpDestroy(solver_handle);
	hipsparseDestroy(sparse_handle);
	return 0;
}

bool cuSpSolver(int* ARow, int* ACol, float* AVal, float* B, float* X, int m, int n, int nnz)
{
	hipsolverStatus_t solver_status;
	hipsparseStatus_t sparse_status;
	hipsolverSpHandle_t solver_handle = 0;
	hipsparseHandle_t sparse_handle = 0;
	hipsparseMatDescr_t descr = 0;

	// initialize
	sparse_status = hipsparseCreate(&sparse_handle);
	if(sparse_status != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreate failed" << endl;
		return 2;
	}
	sparse_status = hipsparseCreateMatDescr(&descr);
	if(sparse_status != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreateMatDescr failed" << endl;
		return 2;
	}
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	solver_status = hipsolverSpCreate(&solver_handle);
	if(solver_status != HIPSOLVER_STATUS_SUCCESS)
	{
		cout << "ERROR: hipsparseCreate failed" << endl;
		return 2;
	}

	int* csrRow = 0; hipMalloc((void**)&csrRow, (m+1)*sizeof(csrRow[0]));
	int* cooRow = 0; hipMalloc((void**)&cooRow, nnz*sizeof(cooRow[0]));

	hipMemcpy(cooRow, ARow, (size_t)(nnz*sizeof(cooRow[0])), hipMemcpyHostToDevice);
	hipsparseXcoo2csr(sparse_handle, cooRow, nnz, m, csrRow, HIPSPARSE_INDEX_BASE_ZERO);
	int AcsrRow[m+1];
	hipMemcpy(AcsrRow, csrRow, (size_t)((m+1)*sizeof(AcsrRow[0])), hipMemcpyDeviceToHost);

	int p[n];// = {0};
	int rankA;
	float min_norm;
	cout<<"go"<<endl;
	solver_status = cusolverSpScsrlsqvqrHost(
		solver_handle,	// handle
		m,
		n,
		nnz,
		descr,
		AVal,
		AcsrRow,
		ACol,
		B,
		0.0,			// tol
		&rankA,			// rankA
		X,				
		p,				// p
		&min_norm);		// min_norm
	if(solver_status != HIPSOLVER_STATUS_SUCCESS)
	{
		cout<<"solve failed"<<endl;
		return 1;
	}

	hipFree(csrRow);
	hipFree(cooRow);
	hipsolverSpDestroy(solver_handle);
	hipsparseDestroy(sparse_handle);
	return 0;
}

// void Result();
